/*
 * MemoryManagement.cpp
 *
 *  Created on: Jul 1, 2018
 */

#include <MemoryManagement.h>
#include <unistd.h>

namespace MemoryManagement
{
	template<typename T>
	void safeFree(T* pointer)
	{
		if(pointer != nullptr && pointer != 0)
		{
			pointer->~T();
			free(pointer);
		}
	}

	template<typename T>
	void safeCUDAFree(T* pointer)
	{
		if(pointer != nullptr && pointer != 0)
		{
			pointer->~T();
			hipFree(pointer);
		}
	}


	template<typename T>
	__host__ __device__
	void copyArrays(T* source, T* destination, sizeType arrayLengths)
	{
		sizeType index;

		for(index = 0; index < arrayLengths; index++)
		{
			destination[index] = source[index];
		}
	}

	template<typename T>
	void transferMemoryToCUDA(T* source, T* destination, sizeType sizeOfTransfer)
	{
		hipMemcpy(destination, source, sizeOfTransfer, hipMemcpyHostToDevice);
	}

	template<typename T>
	void transferMemoryFromCUDA(T* source, T* destination, sizeType sizeOfTransfer)
	{
		hipMemcpy(destination, source, sizeOfTransfer, hipMemcpyDeviceToHost);
	}

	void allocateMemory(void** pointer, unsigned int size, bool isCUDA)
	{
		if(isCUDA)
		{
			hipMalloc(pointer, size);
		}
		else
		{
			(*pointer) = malloc(size);
		}
	}

	template void safeFree<float>(float*);
	template void safeFree<unsigned int>(unsigned int*);

	template void safeCUDAFree<float>(float*);
	template void safeCUDAFree<unsigned int>(unsigned int*);

	template void copyArrays<unsigned int>(unsigned int*, unsigned int*, sizeType);
	template void copyArrays<float>(float*, float*, sizeType);

	template void transferMemoryToCUDA<unsigned int>(unsigned int*, unsigned int*, sizeType);
	template void transferMemoryToCUDA<float>(float*, float*, sizeType);
}
