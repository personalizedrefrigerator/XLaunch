#include "hip/hip_runtime.h"
/*
 * CUDAColorGrapherRenderer.cpp
 *
 *  Created on: Jun 29, 2018
 */

#include "CUDAColorGrapherRenderer.h"
#include <chrono>
#include "TextDrawer.h"
#include <sstream>

__global__ void fillPixels(char * pixelData, int width, int height, float tX, float tY, float zoomX, float zoomY, float time)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int dataX = index % width;
	int dataY = index / width;
	float x = (dataX + tX) * zoomX;
	float y = (dataY + tY) * zoomY;

	if(dataX + dataY*width*4 + 2 < width * height * 4)
	{
		pixelData[dataY*width*4 + dataX*4 + 2] = (sinf(tanf(x*y - x) * cosf(x/(sinf(time/10 + y))) - tanf(y))/((sin(x*x*y-y*tan(time/100))+5)/6) + 1) * 256 / 2;
		pixelData[dataY*width*4 + dataX*4] = 0.5 * ((cosf(x/(y+sin(time/5))) + tanf(sin(time/50 - x) * y / (x+1))) / 2 + 1)*256;
		pixelData[dataY*width*4 + dataX*4 + 1] = 0.5 * (sinf(x*y + tanf(x*x*x - y*y) + cosf(tanf(x/(x+y+sinf(time/2.0)))*cosf(time))) + 1)*256;
	}
}

void CUDAColorGrapherRenderer::render(PixelSetter* pixelSetter, int width, int height)
{
	if(width * height == 0)
	{
		return;
	}

	if(invertedPixelSetter == nullptr)
	{
		invertedPixelSetter = new InvertedPixelSetter(pixelSetter);

		// Multiply the values of red, green, and blue to make text more visible.
		invertedPixelSetter->multiplyRed = 1.0f;
		invertedPixelSetter->multiplyGreen = 0.0f;
		invertedPixelSetter->multiplyBlue = 1.0f;
	}

	invertedPixelSetter->setPixelSetter(pixelSetter);

	std::chrono::steady_clock::time_point startTime = std::chrono::steady_clock::now();
	std::chrono::steady_clock::duration durationSinceEpoch = startTime.time_since_epoch();
	long time = std::chrono::duration_cast<std::chrono::microseconds>(durationSinceEpoch).count();
	//long startTimeSinceEpoch = time; // Explicitly store the starting time.

	time /= 1000; // Convert time to milliseconds.

	long tX = -width/2;
	long tY = -height/2 + sin(time / 5000.0) * 250;
	zoom = (tan(time / 10000.0) + 1) / 100.0;
	float xZoom = zoom * cos(sin(time/1000.0/6.28)/2);

	char * pixelData = pixelSetter->getDataPtr();

	size_t length = width * height * sizeof(float);

	char * devicePixelData;
	hipMalloc(&devicePixelData, length);

	int threadsPerBlock = 256;
	int blocksPerGrid = width * height / threadsPerBlock;
	fillPixels<<<blocksPerGrid, threadsPerBlock>>>(devicePixelData, width, height, tX, tY, xZoom, zoom, time/1000.0f);

	hipDeviceSynchronize();

	hipMemcpy(pixelData, devicePixelData, length, hipMemcpyDeviceToHost);

	hipFree(devicePixelData);

	// Find the elapsed time.
	std::chrono::steady_clock::time_point endTime = std::chrono::steady_clock::now();
	long endTimeSinceEpoch = (std::chrono::duration_cast<std::chrono::microseconds>(endTime.time_since_epoch())).count();
	double millisecondDelta = (endTimeSinceEpoch - lastTimeSinceEpoch) / 1000.0;
	lastTimeSinceEpoch = endTimeSinceEpoch;

	// Calculate the frames per second.
	float framesPerSecond = millisecondDelta;

	if(millisecondDelta != 0)
	{
		framesPerSecond = 1 / (millisecondDelta / 1000.0);
	}

	// Display it.
	std::stringstream outputText;
	outputText << "FPS: " << framesPerSecond << ", size: ";
	outputText << "(" << width << ", " << height << ")";

	pixelSetter->setColor(200, 200, 0);
	textRenderer.draw(outputText.str(), 0, 0, *invertedPixelSetter, 2, 2);
}

CUDAColorGrapherRenderer::~CUDAColorGrapherRenderer()
{
	// Deconstruct all variables
	if(invertedPixelSetter != nullptr)
	{
		delete invertedPixelSetter;
	}
}

